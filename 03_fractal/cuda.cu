#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>



void hipError_t(hipError_t err)
    {    
	if (err != hipSuccess)
	    {
    		fprintf(stderr, "Failed in %s::%d (error code %s)!\n", __FILE__, __LINE__, hipGetErrorString(err));
    		exit(EXIT_FAILURE);
	    };
    }



__global__ void kernelTest(short *img,  float cre, float cim, float dre, float dim, float pre, float pim )
    {

	int i=blockDim.x*blockIdx.x  +  threadIdx.x;
	assert(i>=0);
	int j=blockDim.y*blockIdx.y  +  threadIdx.y;
	assert(j>=0);
	int ind=i*1024+j; // адрес пикселя в массиве
	assert(ind>=0 && ind<1024*1024);
	// координаты пикселя на комплексной плоскости
	double x=cre +  (((double)i)/1024.0-0.5)*dre;
	double y=cim +  (((double)j)/1024.0-0.5)*dim;
	//img[ind]=(short)(x*x+y*y);
	double re=x;
	double im=y;
	double R=1e2;//0.5+0.5*sqrt(1+4*sqrt(cre*cre+cim*cim));
	int cnt=0;
	do {
	double newre=re*re-im*im+pre;
	double newim=re*im+im*re+pim;
	re=newre;
	im=newim;
	cnt++;
//	} while((x-re)*(x-re)+(y-im)*(y-im)<1e4 && cnt<256);
	} while((re)*(re)+(im)*(im)<R*R && cnt<256*4);
	img[ind]=cnt*256/4;
    };



__host__ int fractal(short *img, float cre, float cim, float dre, float dim, float pre, float pim)
    {

	hipError_t err;
	int N=1024;
	int size=N*N*sizeof(short);
	int K=16;
	dim3 blockSize = dim3(K,K,1);
	dim3 gridSize  = dim3(N/K,N/K,1);


	short *dA;

//	cudaSetDevice(0);
	hipError_t( hipGetLastError() );
	hipError_t( hipMalloc(&dA, size) );

	kernelTest <<<gridSize, blockSize>>> (dA, cre, cim, dre, dim, pre, pim );
	hipError_t(hipMemcpy(img, dA, size, hipMemcpyDeviceToHost) );

	hipFree(dA);
	return 0;
    };    
