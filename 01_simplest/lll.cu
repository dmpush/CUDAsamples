#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>

#define TYPE float


void hipError_t(hipError_t err)
    {    
	if (err != hipSuccess)
	    {
    		fprintf(stderr, "Failed in %s::%d (error code %s)!\n", __FILE__, __LINE__, hipGetErrorString(err));
    		exit(EXIT_FAILURE);
	    };
    }



__global__ void kernelTest(TYPE *A)
    {
	int i=blockDim.y*threadIdx.x   +  threadIdx.y;
//	for(int k=0; k<1024*1024*1024; k++)
	    A[i] = i+1;
//	assert(0);
    };

__host__ int main(int argc, char *argv[])
    {
	hipError_t err;
	int N=10;
	int size=N*N*sizeof(TYPE);

	dim3 blockSize = dim3(N,N,1);
	dim3 gridSize  = dim3(1,1,1);

	TYPE *hA;
	hA=(TYPE*)malloc(size);

	for(int n=0; n<N*N; n++)
	    hA[n]=n;

	TYPE *dA;

	hipSetDevice(0);
	hipError_t( hipGetLastError() );
	hipError_t( hipMalloc(&dA, size) );

//	cudaError(cudaMemcpy(dA, hA, size, cudaMemcpyHostToDevice) );

	kernelTest <<<gridSize, blockSize>>> (dA);

/*
	cudaEvent_t syncEvent;
	cudaError( cudaEventCreate(&syncEvent) );
	cudaError( cudaEventRecord(syncEvent, 0) );
	cudaError( cudaEventSynchronize(syncEvent) );
*/
//	cudaError( cudaDeviceSynchronize()  );


	hipError_t(hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost) );

	for(int i=0; i<10; i++)
	    printf("Result: %d %f\n", i, (float)hA[i]);
	printf("Ok!\n");
	hipFree(dA);
	free(hA);
	
	return 0;
    };    
